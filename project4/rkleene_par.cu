
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <math.h>

#define row(idx, N) (idx / N)
#define col(idx, N) (idx % N)

__global__ void shortest_path(int* edges_in, int *edges_out, int N, int* change){
	// multiply edges_in by itself
    int tid, via, src, dst;
    tid = threadIdx.x + blockIdx.x * 1024;
    
    edges_out[tid] = edges_in[tid];
    src = row(tid,N);
    dst = col(tid,N);
    for(via = 0; via < N; via++) {
        if(edges_in[src*N + via] != INT_MAX && edges_in[via*N + dst] != INT_MAX){
            if(edges_out[tid] > (edges_in[src*N + via]+edges_in[via*N + dst])){
                edges_out[tid] = edges_in[src*N + via]+edges_in[via*N + dst];
                *change = 1;
            }
        }
    }
}

int main(int argc, char *argv[])
{      
    int i, src, dst, weight;
    int N;
    int *edges_in, *edges_out;

    // Time
    /*
    time_t start, end;
    double result;
    start = clock();
    */

    if(argc < 3){
        printf("Usage: %s input_file <print>\n", argv[0]);
        exit(1);
    }

    FILE *fp = fopen(argv[1], "r");
    if(fp==NULL) {
        perror("File open error: ");
        exit(1);
    }

    int print_flag = atoi(argv[2]);

    fscanf(fp, "%d", &N);
    edges_in = (int*) malloc(sizeof(int)*N*N);
    edges_out = (int*) malloc(sizeof(int)*N*N);
    for(i=0;i<N*N;i++){
        if(row(i,N) == col(i,N)) {
            edges_in[i] = 0;
	}
	else {
            edges_in[i] = INT_MAX;
	}
    }

    while (fscanf(fp, "%d %d %d\n", &src, &dst, &weight) > 0) {
        edges_in[src*N + dst] = weight;
    }

    // cudaMalloc
    int *edges_in_d, *edges_out_d, *change_d, *tmp;
    int change;
    hipMalloc((void**)&edges_in_d, sizeof(int)*N*N);
    hipMalloc((void**)&edges_out_d, sizeof(int)*N*N);
    hipMalloc((void**)&change_d, sizeof(int));
    
    // cudaMemcpy from host to device
    hipMemcpy(edges_in_d, edges_in, sizeof(int)*N*N, hipMemcpyHostToDevice);

    while(1){
        change = 0;
        hipMemcpy(change_d, &change, sizeof(int), hipMemcpyHostToDevice);
        shortest_path<<<1024,1024>>>(edges_in_d, edges_out_d, N, change_d); 
        hipMemcpy(&change, change_d,sizeof(int), hipMemcpyDeviceToHost);
        if(change){
            tmp = edges_out_d;
            edges_out_d = edges_in_d;
            edges_in_d = tmp;
        }
        else break;
    }

    // cudaMemcpy from device to host 
    hipMemcpy(edges_out, edges_out_d, sizeof(int)*N*N, hipMemcpyDeviceToHost);

    if(print_flag){
        for(i = 0; i < N*N; i++) {
            if(edges_out[i] !=0 && edges_out[i] != INT_MAX){
                printf("%d %d %d\n", row(i,N), col(i,N), edges_out[i]);
            }
        }
    }

    free(edges_in);
    free(edges_out);

    // Time
    /*
    end = clock();
    result = (double)(end - start)/CLOCKS_PER_SEC;
    if(!print_flag)
        printf("%fms\n",result);
    */

    return 0;
}
